#include "hip/hip_runtime.h"
/*
 *  Ejercicio 2 Práctica 3: CUDA
 *  Desempeño en función de la homogeneidad para acceder a memoria
 *  y de la regularidad del código
 */

#include <stdio.h>
//PP#include <hip/hip_runtime.h>

#define STRIDE       32
#define OFFSET        0
#define GROUP_SIZE  512

/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
	int n_elem_per_thread = N / (gridDim.x * blockDim.x);
	int block_start_idx = n_elem_per_thread * blockIdx.x * blockDim.x;
	int thread_start_idx = block_start_idx
	+ (threadIdx.x / STRIDE) * n_elem_per_thread * STRIDE
	+ ((threadIdx.x + OFFSET) % STRIDE);
	int thread_end_idx = thread_start_idx + n_elem_per_thread * STRIDE;
	if(thread_end_idx > N) thread_end_idx = N;
	int group = (threadIdx.x / GROUP_SIZE) & 1;
	for(int idx=thread_start_idx; idx < thread_end_idx; idx+=STRIDE)
	{
		if(!group) a[idx] = a[idx] * a[idx];
		else       a[idx] = a[idx] + a[idx];
	}
}

// main routine that executes on the host
int main(void)
{
	float *a_h, *a_d;  // Pointer to host & device arrays
	const int N = 1<<10;  // Make a big array with 2**N elements
	size_t size = N * sizeof(float);
    
    /* Auxiliares para medir tiempos */
    hipEvent_t start, stop;
    float time;

    a_h = (float *)malloc(size);        // Allocate array on host
	hipMalloc((void **) &a_d, size);   // Allocate array on device
	
    // Initialize host array and copy it to CUDA device
	for (int i=0; i<N; i++)
        a_h[i] = (float)i;

	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");

	// Create timer for timing CUDA calculation
	//PPunsigned int timer = 0;
	//PPcutCreateTimer( &timer );
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    // Set number of threads and blocks
	int n_threads_per_block = 128;//1<<9;  // 512 threads per block
	int n_blocks = 256;//1<<10;  // 1024 blocks

	// Do calculation on device

    hipEventRecord(start,0);
	square_array <<< n_blocks, n_threads_per_block >>> (a_d, N);
	hipDeviceSynchronize();  // Wait for square_array to finish on CUDA

    checkCUDAError("kernel invocation");


	// Retrieve result from device and store it in host array
	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy");

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime( &time, start, stop );

	// Print some of the results
	for (int i=0; i<N; i+=N/50) printf("%d %f\n", i, a_h[i]);

    // Imprime tiempo de ejecución
    printf("\n\nTIEMPO DE EJECUCIÓN: %f mSeg\n\n", time);

    hipEventDestroy( start );
    hipEventDestroy( stop );

	free(a_h); hipFree(a_d);
}

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}